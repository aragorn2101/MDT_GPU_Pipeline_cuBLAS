#include "hip/hip_runtime.h"
/*
 *  CUDA kernel for re-ordering the data before running cuBLAS kernel.
 *
 *  Copyright (C) 2019 Nitish Ragoomundun
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU Affero General Public License as
 *  published by the Free Software Foundation, either version 3 of the
 *  License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU Affero General Public License for more details.
 *
 *  You should have received a copy of the GNU Affero General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *
 */

#include <hip/hip_runtime.h>

/*
 *  GPU kernel to re-order array data
 *
 *  Nelements: number of elements in the array x number of polarisations
 *             this flexibility allows to process for a single
 *             polarisation if needed,
 *  Npol: number of polarisations,
 *  Nchannels: number of frequency channels in each spectrum,
 *  FOutput: array output from F-engine,
 *  XInput: array to be input to cuBLAS kernel.
 *
 *  Example call:
 *
 *  NumThreadx = (Npol*Nelements >= 32) ? 32 : (Npol*Nelements);
 *  NumThready = 32;
 *  NumThreadz = 1;
 *  NumBlockx  = Nspectra;
 *  NumBlocky  = Nchannels/NumThready + ((Nchannels%NumThready != 0) ? 1 : 0);
 *  NumBlockz  = (Npol*Nelements)/NumThreadx + (((Npol*Nelements)%NumThreadx != 0) ? 1 : 0);
 *
 *  ReorderXInput<<< dim3(NumBlockx,NumBlocky,NumBlockz), dim3(NumThreadx,NumThready,NumThreadz) >>>(Npol*Nelements, Nchannels, d_FOutput, d_XInput);
 *
 */
__global__ void ReorderXInput(int Nelements, int Nchannels, hipfftComplex *FOutput, hipComplex *XInput)
{
  __shared__ hipfftComplex sh_Temp[32][32];

  int channelIdx = blockIdx.y*blockDim.y + threadIdx.y;
  int elementIdx = blockIdx.z*blockDim.x + threadIdx.x;
  int FOutputIdx = (blockIdx.x*Nelements + elementIdx)*Nchannels + channelIdx;
  int XInputIdx  = (channelIdx*gridDim.x + blockIdx.x)*Nelements + elementIdx;

  if (channelIdx < Nchannels && elementIdx < Nelements)
  {
    // Copy data from output of F-engine
    sh_Temp[threadIdx.x][threadIdx.y] = FOutput[FOutputIdx];

    // Copy data to input for X-engine
    XInput[XInputIdx] = sh_Temp[threadIdx.x][threadIdx.y];
  }
}
